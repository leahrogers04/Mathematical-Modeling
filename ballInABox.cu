#include "hip/hip_runtime.h"
//nvcc ballInABox.cu -o bounce -lglut -lm -lGLU -lGL																														
//To stop hit "control c" in the window you launched it from.
// This is now on github
#include <iostream>
#include <fstream>
#include <sstream>
#include <string.h>
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
using namespace std;

float TotalRunTime;
float RunTime;
float Dt;
float4 Position, Velocity, Force;
float SphereMass;
float SphereDiameter;
float BoxSideLength;

// Window globals
static int Window;
int XWindowSize;
int YWindowSize;
double Near;
double Far;
double EyeX;
double EyeY;
double EyeZ;
double CenterX;
double CenterY;
double CenterZ;
double UpX;
double UpY;
double UpZ;

void setInitailConditions();
void drawPicture();
void getForces();
void updatePositions();
void nBody();
void startMeUp();

void Display()
{
	drawPicture();
}

void idle()
{
	nBody();
}

void reshape(int w, int h)
{
	glViewport(0, 0, (GLsizei) w, (GLsizei) h);
}

void setInitailConditions()
{
	Position.x = 0.0;
	Position.y = 0.0;
	Position.z = 0.0;
	
	Velocity.x = 60.0;
	Velocity.y = 60.0;
	Velocity.z = 60.0;
	
	Force.x = 0.0;
	Force.y = 0.0;
	Force.z = 0.0;
	
	SphereDiameter = 0.5;
	
	SphereMass = 0.5;
	
	BoxSideLength = 5.0;
	
	TotalRunTime = 10000.0;
	RunTime = 0.0;
	Dt = 0.001;

	
}

void drawPicture()
{
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	
	float halfSide = BoxSideLength/2.0;
	
	glColor3d(1.0,0.2823529, 0.6470588);
	glPushMatrix();
		glTranslatef(Position.x, Position.y, Position.z);
		glutSolidSphere(SphereDiameter/2.0, 30, 30);
	glPopMatrix();
	
	glLineWidth(3.0);
	//drawing front of box
	glColor3d(0.0, 1.0, 0.0);
	glBegin(GL_LINE_LOOP);
		glVertex3f(-halfSide, -halfSide, halfSide);
		glVertex3f(halfSide, -halfSide, halfSide);
		glVertex3f(halfSide, halfSide, halfSide);
		glVertex3f(-halfSide, halfSide, halfSide);
		glVertex3f(-halfSide, -halfSide, halfSide);
	glEnd();
	glEnd();
	//Drawing back of box
	glColor3d(1.0, 1.0, 1.0);
	glBegin(GL_LINE_LOOP);
		glVertex3f(-halfSide, -halfSide, -halfSide);
		glVertex3f(halfSide, -halfSide, -halfSide);
		glVertex3f(halfSide, halfSide, -halfSide);
		glVertex3f(-halfSide, halfSide, -halfSide);
		glVertex3f(-halfSide, -halfSide, -halfSide);
	glEnd();
	// Finishing off right side
	glBegin(GL_LINES);
		glVertex3f(halfSide, halfSide, halfSide);
		glVertex3f(halfSide, halfSide, -halfSide);
		glVertex3f(halfSide, -halfSide, halfSide);
		glVertex3f(halfSide, -halfSide, -halfSide);
	glEnd();
	// Finishing off left side
	glBegin(GL_LINES);
		glVertex3f(-halfSide, halfSide, halfSide);
		glVertex3f(-halfSide, halfSide, -halfSide);
		glVertex3f(-halfSide, -halfSide, halfSide);
		glVertex3f(-halfSide, -halfSide, -halfSide);
	glEnd();
	
	
	glutSwapBuffers();
}

void getForces()
{
	// !!!! you probably should work on these guys/gals.
	Force.x = 50.0;
	Force.y = 50.0;
	Force.z = 50.0;

    float halfSide = BoxSideLength / 2.0;
    float k = 70000.0;  // Very high spring constant to ensure the ball doesn't escape
	float radius = SphereDiameter /2.0;

// this is a spring force that repells the ball when it gets too close to the wall
  
  //
    if(Position.x + radius > halfSide) //this checks if the balls rightmost point passed the right wall
    {
        Force.x -= k * (Position.x + radius - halfSide); //if its true this applies a force in -x to push it back inside
    } 
    else if(Position.x - radius < -halfSide) //this is checking if the balls left most side has passed the left wall
    {
        Force.x -= k * (Position.x - radius + halfSide); //if its true then this applies a force in +x to push it back inside
    }

  
    if(Position.y + radius > halfSide) 
    {
        Force.y -= k * (Position.y + radius - halfSide);
    } 
    else if(Position.y - radius < -halfSide) 
    {
        Force.y -= k * (Position.y - radius + halfSide);
    }

    // Apply force in the z direction
    if(Position.z + radius > halfSide) 
    {
        Force.z -= k * (Position.z + radius - halfSide);
    } 
    else if(Position.z - radius < -halfSide) 
    {
        Force.z -= k * (Position.z - radius + halfSide);
    }
}

   




void updatePositions()
{
	// These are the LeapFrog formulas.
	if(RunTime == 0.0)
	{
		Velocity.x += (Force.x/SphereMass)*(Dt/2.0);
		Velocity.y += (Force.y/SphereMass)*(Dt/2.0);
		Velocity.z += (Force.z/SphereMass)*(Dt/2.0);
	}
	else
	{
		Velocity.x += (Force.x/SphereMass)*Dt;
		Velocity.y += (Force.y/SphereMass)*Dt;
		Velocity.z += (Force.z/SphereMass)*Dt;
	}

	Position.x += Velocity.x*Dt;
	Position.y += Velocity.y*Dt;
	Position.z += Velocity.z*Dt;

	 float halfSide = BoxSideLength / 2.0;
	 float maxVelocity = 60.0f;
	 float radius = SphereDiameter /2.0;


    if(Position.x + radius > halfSide || Position.x - radius < -halfSide)
    {
        Velocity.x += ((rand() % 21) - 10) * 0.05f;  // adding a random component so that the ball doesnt just go from corner to corner
		Velocity.x = (Velocity.x > maxVelocity) ? maxVelocity : (Velocity.x < -maxVelocity) ? -maxVelocity : Velocity.x; // this makes sure it doesnt go over the max velocity so that it doesnt shoot off into oblivian.

    }

    if(Position.y + radius > halfSide || Position.y - radius < -halfSide)
    {
        Velocity.y += ((rand() % 21) - 10) * 0.05f;  // Add small random component
		Velocity.y = (Velocity.y > maxVelocity) ? maxVelocity : (Velocity.y < -maxVelocity) ? -maxVelocity : Velocity.y;

    }

    if(Position.z + radius > halfSide || Position.z - SphereDiameter/2.0 < -halfSide)
    {
        Velocity.z += ((rand() % 21) - 10) * 0.05f;  // Add small random component
		Velocity.z = (Velocity.z > maxVelocity) ? maxVelocity : (Velocity.z < -maxVelocity) ? -maxVelocity : Velocity.z;

    }
}


void nBody()
{	
	getForces();
	updatePositions();
	drawPicture();
	printf("\n Time = %f", RunTime);
	RunTime += Dt;
	
	if(TotalRunTime < RunTime)
	{
		glutDestroyWindow(Window);
		printf("\n Later Dude \n");
		exit(0);
	}
}

void startMeUp() 
{	
	// The Rolling Stones
	// Tattoo You: 1981
	setInitailConditions();
}

int main(int argc, char** argv)
{
	startMeUp();
	
	XWindowSize = 1000;
	YWindowSize = 1000; 

	// Clip plains
	Near = 0.2;
	//Far = BoxSideLength;
	Far = 10.0;

	//Where your eye is located
	EyeX = 0.0;
	EyeY = 0.0;
	EyeZ = 6.0;

	//Where you are looking
	CenterX = 0.0;
	CenterY = 0.0;
	CenterZ = 0.0;

	//Up vector for viewing
	UpX = 0.0;
	UpY = 1.0;
	UpZ = 0.0;
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(5,5);
	Window = glutCreateWindow("Particle In A Box");
	
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
	glMatrixMode(GL_MODELVIEW);
	
	glClearColor(0.0, 0.0, 0.0, 0.0);
		
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	//glutMouseFunc(mymouse);
	//glutKeyboardFunc(KeyPressed);
	glutIdleFunc(idle);
	glutMainLoop();
	
	return 0;
}
